#include "hip/hip_runtime.h"
//
// Created by bjgilhul on 1/5/23.
//

#include "visibility.h"
#include <float.h>
#include <iostream>

namespace polycheck {

    __device__ double
    line_observation( const double* data, int height, int width, int sx, int sy, int ex, int ey ) {
        // Using Bresenham implementation found at:
        //   http://members.chello.at/~easyfilter/Bresenham.pdf
        auto dx = abs(sx-ex);
        auto step_x = sx < ex ? 1 : -1;
        auto dy = -abs(sy-ey);
        auto step_y = sy < ey ? 1 : -1;
        auto error = dx + dy;

        auto observation = 1.0;    // assume the point is initially viewable
        for( ;; ) {
            observation *= (1.0 - data[ sy * width + sx]);
            if( observation < FLT_EPSILON*2 ) {
                break;
            }
            auto e2 = 2 * error;
            if( e2 >= dy ) {
                if( sx == ex ) {
                    break;
                }
                error = error + dy;
                sx += step_x;
            }
            if( e2 <= dx ) {
                if( sy == ey ) {
                    break;
                }
                error += dx;
                sy += step_y;
            }
        }

        return observation;
    }

    __global__ void
    check_visibility(const double *data, const int height, const int width, const int *start,
                          const int *ends, int num_ends, double *results ) {

        auto start_index = blockIdx.x * blockDim.x + threadIdx.x;
        auto stride = blockDim.x * gridDim.x;

        for (auto i = start_index; i < num_ends; i += stride) {
            int ex, ey;
            if( ends == nullptr ) {
                ex = int(i % width);
                ey = int(i / width);
            } else {
                ex = ends[i*2];
                ey = ends[i*2+1];
            }
            results[ey*width + ex] = line_observation( data, height, width, start[0], start[1], ex, ey );
        }
    }

    __global__ void
    check_region_visibility(const double *data, const int height, const int width, const int *starts, int num_starts,
                          const int *ends, int num_ends, double *results ) {

        auto ends_index = blockIdx.x * blockDim.x + threadIdx.x;
        auto ends_stride = blockDim.x * gridDim.x;
        auto starts_index = blockIdx.y * blockDim.y + threadIdx.y;
        auto starts_stride = blockDim.y * gridDim.y;

        for (auto si = starts_index; si < num_starts; si += starts_stride) {
            auto sx = starts[si*2];
            auto sy = starts[si*2+1];

            for (auto ei = ends_index; ei < num_ends; ei += ends_stride) {
                auto ex = ends[ei*2];
                auto ey = ends[ei*2+1];
                results[si * num_ends + ei] = line_observation(data, height, width, sx, sy, ex, ey);
            }
        }
    }


    void
    visibility( const double* data, int height, int width, double* results, const int* start, const int* ends, int num_ends ) {

        double *cuda_data;
        double *cuda_result;
        int *cuda_ends;
        int *cuda_start;

        auto data_size = height * width * sizeof(double);
        auto start_size = 2 * sizeof(int);
        auto ends_size = num_ends * start_size;

        CUDA_CALL(hipMalloc( &cuda_data, data_size));
        CUDA_CALL(hipMemcpy( cuda_data, data, data_size, hipMemcpyHostToDevice));
        CUDA_CALL(hipMalloc( &cuda_start, start_size));
        CUDA_CALL(hipMemcpy( cuda_start, start, start_size, hipMemcpyHostToDevice));
        if( num_ends > 0 ) {
            CUDA_CALL(hipMalloc( &cuda_ends, ends_size));
            CUDA_CALL(hipMemcpy( cuda_ends, ends, ends_size, hipMemcpyHostToDevice));
        } else {
            cuda_ends = nullptr;
            num_ends = height * width;
        }
        CUDA_CALL(hipMalloc( &cuda_result, data_size));
        CUDA_CALL(hipMemset(cuda_result, 0, data_size));

        auto block_size = BLOCK_SIZE;
        auto num_blocks = std::max(MAX_BLOCKS, int((num_ends + block_size - 1) / block_size));
        check_visibility<<<num_blocks, block_size>>>(cuda_data, height, width, cuda_start, cuda_ends, num_ends, cuda_result);

        // copy the results back from the device
        CUDA_CALL(hipMemcpy(results, cuda_result, height * width * sizeof(double), hipMemcpyDeviceToHost));

        // release the memory
        CUDA_CALL(hipFree(cuda_data));
        CUDA_CALL(hipFree(cuda_start));
        CUDA_CALL(hipFree(cuda_result));
        if( cuda_ends != nullptr){
            CUDA_CALL(hipFree(cuda_ends));
        }
    }


    void
    visibility_from_region( const double* data, int height, int width, double* results, const int* starts,
                            int num_starts, const int* ends, int num_ends ) {

        double *cuda_data;
        double *cuda_result;
        int *cuda_ends;
        int *cuda_start;

        auto data_size = height * width * sizeof(double);
        auto start_size = num_starts * 2 * sizeof(int);
        auto ends_size = num_ends * 2 * sizeof(int);

        CUDA_CALL(hipMalloc( &cuda_data, data_size));
        CUDA_CALL(hipMemcpy( cuda_data, data, data_size, hipMemcpyHostToDevice));
        CUDA_CALL(hipMalloc( &cuda_start, start_size));
        CUDA_CALL(hipMemcpy( cuda_start, starts, start_size, hipMemcpyHostToDevice));
        CUDA_CALL(hipMalloc( &cuda_ends, ends_size));
        CUDA_CALL(hipMemcpy( cuda_ends, ends, ends_size, hipMemcpyHostToDevice));

        // The space required for results is now a value for each desired end-point, arranged in rows for each
        // desired starting point. NOTE: this can be very large...
        auto results_size = num_starts * num_ends * sizeof(double);
        CUDA_CALL(hipMalloc( &cuda_result, results_size));
        CUDA_CALL(hipMemset(cuda_result, 0, results_size));

        std::cout << "Calling region visibility check with " << num_starts << " starts and " << num_ends << " ends." << std::endl;

        auto x_block_size = BLOCK_SIZE / Y_BLOCK_SIZE;
        dim3 block( x_block_size, Y_BLOCK_SIZE);
        dim3 grid( std::max( 1, std::min(MAX_BLOCKS, int((num_ends + x_block_size - 1) / x_block_size))),
                   std::max( 1, std::min(MAX_BLOCKS, int((num_starts + Y_BLOCK_SIZE - 1) / Y_BLOCK_SIZE))));

        std::cout << "Using a grid of size " << grid.x << "," << grid.y << " and blocks of " << block.x  << "," << block.y << std::endl;

        check_region_visibility<<<grid, block>>>(cuda_data, height, width,
                                                 cuda_start, num_starts,
                                                 cuda_ends, num_ends, cuda_result);

        // copy the results back from the device
        CUDA_CALL(hipMemcpy(results, cuda_result, results_size, hipMemcpyDeviceToHost));

        // release the memory
        CUDA_CALL(hipFree(cuda_data));
        CUDA_CALL(hipFree(cuda_start));
        CUDA_CALL(hipFree(cuda_result));
        if( cuda_ends != nullptr){
            CUDA_CALL(hipFree(cuda_ends));
        }
   }



}